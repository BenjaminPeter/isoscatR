
#include <hip/hip_runtime.h>
__global__ void powered_exponential_kernel(double* dist, double* cov,
                                           const int n, const int nm, 
                                           const double sigma2, const double phi, 
                                           const double kappa, const double nugget) 
{
    int n_threads = gridDim.x * blockDim.x;
    int pos = blockDim.x * blockIdx.x + threadIdx.x;

    for (int i = pos; i < nm; i += n_threads)
        cov[i] = sigma2 * exp( -pow(dist[i] / phi, kappa) ) + nugget*( i%n == 0 );
    
}


void cov_powered_exponential_gpu(double* dist, double* cov,
                                 const int n, const int m,
                                 double sigma2, double phi, 
                                 double kappa, double nugget,
                                 int n_threads) 
{
    int nm = n*m;
    int blocks = (n+n_threads-1)/n_threads;
    
    powered_exponential_kernel<<<blocks, n_threads>>>(dist, cov, n, nm, sigma2, phi, kappa, nugget);
}
